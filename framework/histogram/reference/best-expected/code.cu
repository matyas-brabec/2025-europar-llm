#include "hip/hip_runtime.h"


#include <cstdint>

constexpr unsigned int itemsPerThread = 256;
constexpr unsigned int privCopies = 32;


template<typename T = char, typename RES = unsigned int>
__global__ void hist(const char *data, unsigned int N, unsigned int *result, int fromValue, int toValue)
{
    const unsigned int histogramSize = toValue - fromValue + 1;

    // initialize shared memory
    extern __shared__ unsigned int resultShm[];
    for (unsigned int i = threadIdx.x; i < histogramSize * privCopies; i += blockDim.x) {
        resultShm[i] = 0;
    }

    unsigned int *resultShmPriv = resultShm + threadIdx.x % privCopies;
    __syncthreads();

    // aggregate histograms in shared memory
	unsigned int idx = blockIdx.x * blockDim.x * itemsPerThread + threadIdx.x;
    unsigned int endIdx = min(idx + itemsPerThread*blockDim.x, N);
    while (idx < endIdx) {
        int c = (int)data[idx] - fromValue;
        if (c >= 0 && c < histogramSize) {
            atomicAdd(&resultShmPriv[c * privCopies], 1);
        }
        idx += blockDim.x;
    }

    __syncthreads();

    // merge shared memory histograms
    for (unsigned int i = threadIdx.x; i < histogramSize; i += blockDim.x) {
        unsigned int sum = 0;
        for (unsigned int j = 0; j < privCopies; ++j) {
            sum += resultShm[i * privCopies + j];
        }
        if (sum > 0) {
            atomicAdd(&result[i], sum);
        }
    }
}


void run_histogram(const char *input, unsigned int *histogram, unsigned int inputSize, int from, int to) {
	constexpr unsigned int blockSize = 1024;
    unsigned int threads = inputSize / itemsPerThread;
    unsigned int shmSize = (to - from + 1) * privCopies * sizeof(unsigned int);
	hist<<<(threads + blockSize - 1) / blockSize, blockSize, shmSize>>>(input, inputSize, histogram, from, to);
}
